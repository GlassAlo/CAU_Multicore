#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define SPHERES 20
#define INF     2e10f
#define DIM     2048
#define rnd(x)  (x * rand() / RAND_MAX)

struct Sphere {
    float r, g, b;
    float radius;
    float x, y, z;

    __device__ float hit(float ox, float oy, float* n) const {
        float dx = ox - x;
        float dy = oy - y;
        float rr = radius * radius;
        float d2 = dx * dx + dy * dy;
        if (d2 < rr) {
            float dz = sqrtf(rr - d2);
            *n = dz / radius;
            return dz + z;
        }
        return -INF;
    }
};

// Move spheres to constant memory for faster cached access
__constant__ Sphere d_spheres[SPHERES];

__global__ void render_kernel(unsigned char* ptr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= DIM || y >= DIM) return;

    int offset = x + y * DIM;
    float ox = x - DIM * 0.5f;
    float oy = y - DIM * 0.5f;

    float r = 0, g = 0, b = 0;
    float maxz = -INF;

    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = d_spheres[i].hit(ox, oy, &n);
        if (t > maxz) {
            maxz = t;
            r = d_spheres[i].r * n;
            g = d_spheres[i].g * n;
            b = d_spheres[i].b * n;
        }
    }

    // Use __saturatef to clamp color values and avoid branching
    ptr[offset * 4 + 0] = (unsigned char)(__saturatef(r) * 255.0f);
    ptr[offset * 4 + 1] = (unsigned char)(__saturatef(g) * 255.0f);
    ptr[offset * 4 + 2] = (unsigned char)(__saturatef(b) * 255.0f);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(const unsigned char* bitmap, int xdim, int ydim, FILE* fp) {
    fprintf(fp, "P3\n%d %d\n255\n", xdim, ydim);
    for (int y = 0; y < ydim; y++) {
        for (int x = 0; x < xdim; x++) {
            int i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(int argc, char* argv[]) {
    char *filename = "result.ppm";
    if (argc > 1) {
        filename = argv[1];
    }
    srand(time(NULL));

    Sphere spheres_h[SPHERES];
    for (int i = 0; i < SPHERES; i++) {
        spheres_h[i].r = rnd(1.0f);
        spheres_h[i].g = rnd(1.0f);
        spheres_h[i].b = rnd(1.0f);
        spheres_h[i].x = rnd(2000.0f) - 1000;
        spheres_h[i].y = rnd(2000.0f) - 1000;
        spheres_h[i].z = rnd(2000.0f) - 1000;
        spheres_h[i].radius = rnd(200.0f) + 40;
    }

    // Copy host spheres to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_spheres), spheres_h, sizeof(Sphere) * SPHERES);

    unsigned char *bitmap_h = nullptr;
    unsigned char *bitmap_d = nullptr;
    // Allocate device memory for image buffer
    hipMalloc((void**)&bitmap_d, DIM * DIM * 4);
    // Allocate pinned (page-locked) host memory for image buffer
    hipHostMalloc((void**)&bitmap_h, DIM * DIM * 4, hipHostMallocDefault);

    // Use 16x16 threads per block (256 threads)
    dim3 threads(16, 16);
    // Compute grid size to cover entire image
    dim3 blocks((DIM + threads.x - 1) / threads.x, (DIM + threads.y - 1) / threads.y);

    clock_t start = clock();

    render_kernel<<<blocks, threads>>>(bitmap_d);

    clock_t stop = clock();
    double duration = ((double) (stop - start)) / CLOCKS_PER_SEC;

    // Ensure all threads are finished
    hipDeviceSynchronize();

    // Copy rendered image from device to host
    hipMemcpy(bitmap_h, bitmap_d, DIM * DIM * 4, hipMemcpyDeviceToHost);

    FILE* fp = fopen(filename, "w");
    if (fp) {
        ppm_write(bitmap_h, DIM, DIM, fp);
        fclose(fp);
    }

    printf("CUDA ray tracing: %f sec\n", duration);
    printf("[%s] was generated.\n", filename);

    hipFree(bitmap_d);
    hipHostFree(bitmap_h);
    return 0;
}